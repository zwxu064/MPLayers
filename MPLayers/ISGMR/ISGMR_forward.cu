#include "hip/hip_runtime.h"
#include "ISGMR.h"
#include "commonCUDA.cuh"

/*
 * Note
 * Bug 1: 20190803, PyTorch 0.4.1, torch.max and torch.min return min_indices
 *        while pytorch 1.1.0 return max_indices, using PyTorch 1.1.0 now
*/

#ifdef __cplusplus
  extern "C" {
#endif

__global__ void TestLoop(const Param param,
                         int required,
                         float* out) {
  int n_disp = param.n_disp;
  int tid = blockDim.x * blockIdx.x + threadIdx.x;
  if (tid >= required) return;

  int value = 99;
  for (int d = 0; d < n_disp; ++d) {
    if (d == 0) {
      value = 2;
//      out[tid] = 2;
    } else if (d == 15) {
      value = 15;
    }
//    printf("value=%d.\n", value);
  }
  out[tid] = value;
}

__global__ void CalLabelKernel(const Param param,
                               const uint n_thread_required,
                               float* cost_final,
                               uchar* label_all) {
  uint tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid >= n_thread_required) return;

  uint n_disp = param.n_disp;
  float min_value = cost_final[tid * n_disp];
  uchar min_index = 0;

  for (uint disp = 1; disp < n_disp; ++disp) {
    float value = cost_final[tid * n_disp + disp];

    if (value < min_value) {
      min_value = value;
      min_index = disp;
    }
  }

  label_all[tid] = min_index;
}

__device__ float MsgSumAll(const int n_dir,
                           const int dir,
                           const int msg_dir_size,
                           const int msg_offset,
                           float* msg) {
  float msg_sum = 0;
  for (int d = 0; d < n_dir; ++d)
    if (d != dir)
      msg_sum += msg[d * msg_dir_size + msg_offset];
  return msg_sum;
}

__global__ void CostAggregateKernel(const bool enable_sgm,
                                    const Param param,
                                    const uint n_thread_required,
                                    float* msg_ptr,
                                    float* cost_final_ptr) {
  uint tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid >= n_thread_required) return;

  uint msg_offset = n_thread_required, n_dir = param.n_dir;
  float value_factor = (float)1;
  if (enable_sgm) value_factor = (float)n_dir;
  float value = cost_final_ptr[tid] * value_factor;

  if (param.enable_min_a_dir) {
    float min_msg = msg_ptr[tid];

    for (uint dir = 1; dir < n_dir; ++dir) {
      float msg = msg_ptr[dir * msg_offset + tid];
      if (msg < min_msg) min_msg = msg;
    }

    value += min_msg;
  } else {
    for (uint dir = 0; dir < n_dir; ++dir) {
      value += msg_ptr[dir * msg_offset + tid] / param.dir_weight;
    }
  }

  cost_final_ptr[tid] = value;
  __syncthreads();
}

__global__ void UpdateUnaryKernel(const Param param,
                                  int n_thread_required,
                                  float* unary_ptr,
                                  float* msg_ptr,
                                  float* unary_update_ptr) {
  uint tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid >= n_thread_required) return;

  uint msg_offset = n_thread_required;
  uint dir_current = param.dir, dir_inv = param.dir_inv, n_dir = param.n_dir;
  float rho = param.rho, value = unary_ptr[tid];

  for (uint dir = 0; dir < n_dir; ++dir)
    value += msg_ptr[dir * msg_offset + tid];

  value -= msg_ptr[dir_current * msg_offset + tid];
  value *= rho;
  value -= msg_ptr[dir_inv * msg_offset + tid];
  unary_update_ptr[tid] = value;
  __syncthreads();
}

__global__ void HorizontalKernel(const Param param,
                                 const uint n_thread_required,
                                 const uint n_thread_a_tree,
                                 float* unary_update,
                                 float* context,
                                 float* edge_weights,
                                 float* msg,
                                 float* msg_update,
                                 uchar* msg_min_index,
                                 uchar* msg_norm_index) {
  static __shared__ float msg_update_shared[MAX_DISPARITY];
  msg_update_shared[threadIdx.x] = 0;
  __syncthreads();

  uint height = param.height, width = param.width;
  uint n_disp = param.n_disp, n_trees = param.n_trees;
  float rho = param.rho;
  int h_step = param.h_step, w_step = param.w_step;
  uint tid = blockIdx.x * blockDim.x + threadIdx.x;  // batch*cv*h*n_thread_a_tree
  uint current_d = threadIdx.x % n_thread_a_tree;
  bool enable_seg = (n_disp == 21);

  if (tid >= n_thread_required) return;
  if (current_d >= n_disp) return;

  uint unary_base = tid / (n_trees * n_thread_a_tree) * height * width * n_disp;
  uint tree_id = (tid / n_thread_a_tree) % n_trees;
  int h_start = tree_id, w_start = (w_step > 0) ? 0 : (width - 1);
  uint edge_base = tid / (n_trees * n_thread_a_tree) * height * width;

  for (uint i = 0; i < width; ++i) {
    int current_node_h = h_start;
    int current_node_w = w_start + i * w_step;
    int front_node_h = current_node_h - h_step;
    int front_node_w = current_node_w - w_step;

    if (0 <= current_node_w && current_node_w < width &&
        0 <= front_node_w && front_node_w < width) {
      float min_value = 0;
      uchar min_idx = 0;
      uint offset_base = unary_base + front_node_h * width * n_disp + front_node_w * n_disp;
      float edge_weight = edge_weights[edge_base + current_node_h * width + current_node_w];

#if TORCH_VERSION_MAJOR == 0
      for (int front_d = 0; front_d < n_disp; ++front_d) {
#else
      for (int front_d = n_disp - 1; front_d >= 0; --front_d) {
#endif
        float context_value = 0;
        if (enable_seg)
         context_value = context[min(current_d, front_d) * n_disp + max(current_d, front_d)];
        else
         context_value = context[std::abs(int(current_d) - int(front_d))];

        uint offset = offset_base + front_d;
        float msg_update_value = rho * msg_update_shared[front_d];
        __syncthreads();
        float value = unary_update[offset] + msg_update_value + edge_weight * context_value;

#if TORCH_VERSION_MAJOR == 0
        if (front_d == 0) {
#else
        if (front_d == n_disp - 1) {
#endif
          min_value = value;
          min_idx = front_d;
        } else if (value < min_value) {
          min_value = value;
          min_idx = front_d;
        }
      }

      msg_update_shared[current_d] = min_value;
      __syncthreads();

      int msg_offset = unary_base + current_node_h * width * n_disp + current_node_w * n_disp + current_d;
      int msg_index_offset = tid / n_thread_a_tree * width + current_node_w;
      uchar norm_idx = 0;

#ifdef USE_MSGNORM_NAIVE
      MsgNormNaive(param.n_disp, current_d, msg_update_shared, &norm_idx);
#else
      MsgNorm(param.n_disp, current_d, msg_update_shared, &norm_idx);
#endif

      msg_update[msg_offset] = msg_update_shared[current_d];

      if (param.is_training) {
        msg_norm_index[msg_index_offset] = norm_idx;
        msg_min_index[msg_offset] = min_idx;
      }
      __syncthreads();
    }
  }
}

__global__ void DiagonalKernelNarrow(const Param param,
                                     const uint n_thread_required,
                                     const uint n_thread_a_tree,
                                     float* unary_update,
                                     float* context,
                                     float* edge_weights,
                                     float* msg,
                                     float* msg_update,
                                     uchar* msg_min_index,
                                     uchar* msg_norm_index) {
  static __shared__ float msg_update_shared[MAX_DISPARITY];
  msg_update_shared[threadIdx.x] = 0;
  __syncthreads();

  uint height = param.height, width = param.width;
  uint n_disp = param.n_disp, n_trees = param.n_trees;
  float rho = param.rho;
  int h_step = param.h_step, w_step = param.w_step;
  uint tid = blockIdx.x * blockDim.x + threadIdx.x;  // batch*cv*n_trees*n_thread_a_tree
  uint current_d = threadIdx.x % n_thread_a_tree;
  uint h_step_abs = std::abs(h_step);
  bool enable_seg = (n_disp == 21);

  if (tid >= n_thread_required) return;
  if (current_d >= n_disp) return;

  uint unary_base = tid / (n_trees * n_thread_a_tree) * height * width * n_disp;
  uint tree_id = (tid / n_thread_a_tree) % n_trees;
  int tree_id_shift = tree_id - (height - 1) * max(w_step, 0);
  int common1 = tree_id_shift % h_step_abs;
  float common2 = float(tree_id_shift) / float(h_step_abs);  // This must be float NOT int, will affect ceilf and floorf
  int h_start = 0, w_start = 0;
  uint edge_base = tid / (n_thread_a_tree * n_trees) * height * width;

  // Use a common mode to calculate start points for shortest chains, read my notes for clarity
  if (w_step > 0) {
    h_start = (h_step_abs - common1) % h_step_abs;
    w_start = ceilf(common2);
  } else {
    h_start = common1;
    w_start = floorf(common2);
  }

  if (h_step < 0) h_start = height - 1 - h_start;
  uint roll_step = (height - 1) / h_step_abs;

  for (uint i = 0; i <= roll_step; ++i) {
    int current_node_h = h_start + i * h_step;
    int current_node_w = w_start + i * w_step;
    int front_node_h = current_node_h - h_step;
    int front_node_w = current_node_w - w_step;

    if (0 <= current_node_h && current_node_h < height &&
        0 <= current_node_w && current_node_w < width &&
        0 <= front_node_h && front_node_h < height &&
        0 <= front_node_w && front_node_w < width) {
      float min_value = 0;
      uchar min_idx = 0;
      uint offset_base = unary_base + front_node_h * width * n_disp + front_node_w * n_disp;
      float edge_weight = edge_weights[edge_base + current_node_h * width + current_node_w];

#if TORCH_VERSION_MAJOR == 0
      for (int front_d = 0; front_d < n_disp; ++front_d) {
#else
      for (int front_d = n_disp - 1; front_d >= 0; --front_d) {
#endif
        float context_value = 0;
        if (enable_seg)
          context_value = context[min(current_d, front_d) * n_disp + max(current_d, front_d)];
        else
          context_value = context[std::abs(int(current_d) - int(front_d))];

        uint offset = offset_base + front_d;
        float msg_update_value = rho * msg_update_shared[front_d];
        __syncthreads();
        float value = unary_update[offset] + msg_update_value + edge_weight * context_value;

#if TORCH_VERSION_MAJOR == 0
        if (front_d == 0) {
#else
        if (front_d == n_disp - 1) {
#endif
          min_value = value;
          min_idx = front_d;
        } else if (value < min_value) {
          min_value = value;
          min_idx = front_d;
        }
      }

      msg_update_shared[current_d] = min_value;
      __syncthreads();

      uint msg_offset = unary_base + current_node_h * width * n_disp + current_node_w * n_disp + current_d;
      uint msg_index_offset = tid / (n_thread_a_tree * n_trees) * height * width + current_node_h * width + current_node_w;
      uchar norm_idx = 0;

#ifdef USE_MSGNORM_NAIVE
      MsgNormNaive(param.n_disp, current_d, msg_update_shared, &norm_idx);
#else
      MsgNorm(param.n_disp, current_d, msg_update_shared, &norm_idx);
#endif

      msg_update[msg_offset] = msg_update_shared[current_d];

      if (param.is_training) {
        msg_norm_index[msg_index_offset] = norm_idx;
        msg_min_index[msg_offset] = min_idx;
      }
      __syncthreads();
    }
  }
}

__global__ void DiagonalKernelWide(const Param param,
                                   const uint n_thread_required,
                                   const uint n_thread_a_tree,
                                   float* unary_update,
                                   float* context,
                                   float* edge_weights,
                                   float* msg,
                                   float* msg_update,
                                   uchar* msg_min_index,
                                   uchar* msg_norm_index) {
  static __shared__ float msg_update_shared[MAX_DISPARITY];
  msg_update_shared[threadIdx.x] = 0;
  __syncthreads();

  uint height = param.height, width = param.width;
  uint n_disp = param.n_disp, n_trees = param.n_trees;
  float rho = param.rho;
  int h_step = param.h_step, w_step = param.w_step;
  uint tid = blockIdx.x * blockDim.x + threadIdx.x;  // batch*cv*n_trees*n_thread_a_tree
  uint current_d = threadIdx.x % n_thread_a_tree;
  bool enable_seg = (n_disp == 21);

  if (tid >= n_thread_required) return;
  if (current_d >= n_disp) return;

  uint unary_base = tid / (n_trees * n_thread_a_tree) * height * width * n_disp;
  uint tree_id = (tid / n_thread_a_tree) % n_trees;
  int tree_id_shift = tree_id - (height - 1) * max(w_step, 0);
  uint h_step_abs = std::abs(h_step), roll_step = (height - 1) / h_step_abs;
  int h_start = (h_step > 0) ? 0 : (height - 1), w_start = tree_id_shift;
  uint edge_base = tid / (n_thread_a_tree * n_trees) * height * width;

  for (uint i = 0; i <= roll_step; ++i) {
    int current_node_h = h_start + i * h_step;
    int current_node_w = w_start + i * w_step;
    int front_node_h = current_node_h - h_step;
    int front_node_w = current_node_w - w_step;

    if (0 <= current_node_h && current_node_h < height &&
        0 <= current_node_w && current_node_w < width &&
        0 <= front_node_h && front_node_h < height &&
        0 <= front_node_w && front_node_w < width) {
      float min_value = 0;
      uchar min_idx = 0;
      uint offset_base = unary_base + front_node_h * width * n_disp + front_node_w * n_disp;
      float edge_weight = edge_weights[edge_base + current_node_h * width + current_node_w];

#if TORCH_VERSION_MAJOR == 0
      for (int front_d = 0; front_d < n_disp; ++front_d) {
#else
      for (int front_d = n_disp - 1; front_d >= 0; --front_d) {
#endif
        float context_value = 0;
        if (enable_seg)
          context_value = context[min(current_d, front_d) * n_disp + max(current_d, front_d)];
        else
          context_value = context[std::abs(int(current_d) - int(front_d))];

        uint offset = offset_base + front_d;
        float msg_update_value = rho * msg_update_shared[front_d];
        __syncthreads();
        float value = unary_update[offset] + msg_update_value + edge_weight * context_value;

#if TORCH_VERSION_MAJOR == 0
        if (front_d == 0) {
#else
        if (front_d == n_disp - 1) {
#endif
          min_value = value;
          min_idx = front_d;
        } else if (value < min_value) {
          min_value = value;
          min_idx = front_d;
        }
      }

      msg_update_shared[current_d] = min_value;
      __syncthreads();

      uint msg_offset = unary_base + current_node_h * width * n_disp + current_node_w * n_disp + current_d;
      uint msg_index_offset = tid / (n_thread_a_tree * n_trees) * height * width + current_node_h * width + current_node_w;
      uchar norm_idx = 0;

#ifdef USE_MSGNORM_NAIVE
      MsgNormNaive(param.n_disp, current_d, msg_update_shared, &norm_idx);
#else
      MsgNorm(param.n_disp, current_d, msg_update_shared, &norm_idx);
#endif

      msg_update[msg_offset] = msg_update_shared[current_d];

      if (param.is_training) {
        msg_norm_index[msg_index_offset] = norm_idx;
        msg_min_index[msg_offset] = min_idx;
      }
      __syncthreads();
    }
  }
}

void ForwardCUDA(const bool enable_sgm,
                 const int sgm_single_mode,
                 const float rho,
                 const int n_iter,
                 const bool enable_min_a_dir,
                 const at::Tensor unary,
                 const at::Tensor context,
                 const at::Tensor edge_weights,
                 at::Tensor msg,
                 at::Tensor cost_final,
                 at::Tensor msg_min_index,
                 at::Tensor msg_norm_index,
                 at::Tensor unary_update,
                 at::Tensor msg_update,
                 at::Tensor label_all) {
  uint n_dir = msg.size(0);
  const uint batch = msg.size(1);
  const uint n_cv = msg.size(2);
  const uint height = msg.size(3);
  const uint width = msg.size(4);
  const uint n_disp = msg.size(5);
  float* unary_ptr = unary.data<float>();
  float* context_ptr = context.data<float>();
  float* edge_weight_ptr = edge_weights.data<float>();
  float* msg_ptr = msg.data<float>();  // (n_dir,batch,cv,h,w,n_disp)
  float* cost_final_ptr = cost_final.data<float>();  // (batch,cv,h,w,n_disp)
  uchar* msg_min_index_ptr = nullptr;
  uchar* msg_norm_index_ptr = nullptr;
  float* unary_update_ptr = unary_update.data<float>();
  float* msg_update_ptr = msg_update.data<float>();  // (n_dir,batch,cv,h,w,n_disp)
  uchar* label_all_ptr = nullptr;
  uint n_thread_a_tree = GetNumThreadATree(n_disp, WARP_SIZE);
  bool is_training = msg_min_index.size(0) == 0 ? false : true;
  bool is_backward = false;
  bool enable_cal_label = label_all.size(0) == 0 ? false : true;
  bool enable_sgm_single_dir = sgm_single_mode >= 0 ? true : false;

  if (enable_sgm_single_dir) n_dir = 1;

  // if (enable_sgm) {
  //   printf("Error!!! CUDA version does not support SGM but ISGMR.\n");
  //   return;
  // }

  if (is_training) {
    msg_min_index_ptr = msg_min_index.data<uchar>();  // (n_iter,n_dir,batch,cv,h,w,n_disp)
    msg_norm_index_ptr = msg_norm_index.data<uchar>();  // (n_iter,n_dir,batch,cv,h,w)
  }

  if (enable_cal_label) label_all_ptr = label_all.data<uchar>();

  // Using separate addresses for msg and index
  std::vector<float*> msg_update_address(n_dir), edge_weight_address(n_dir);
  std::vector<uchar*> msg_min_index_address(n_dir), msg_norm_index_address(n_dir);
  std::vector<Param> param_list;
  uint msg_min_size = batch * n_cv * height * width * n_disp;
  uint msg_min_index_size = n_dir * msg_min_size;
  uint msg_norm_size = msg_min_size / n_disp;
  uint msg_norm_index_size = n_dir * msg_norm_size;
  uint n_thread_unary = min(MAX_THREADS_PER_BLOCK, msg_min_size);
  uint n_block_unary = (msg_min_size + n_thread_unary - 1) / n_thread_unary;
//  hipStream_t streams[1];
//  hipStreamCreate(&streams[0]);

  for (uint dir = 0; dir < n_dir; ++dir) {
    msg_update_address[dir] = msg_update_ptr + dir * msg_min_size;
    edge_weight_address[dir] = edge_weight_ptr + dir * msg_norm_size;
    msg_min_index_address[dir] = nullptr;
    msg_norm_index_address[dir] = nullptr;
    uint dir_actual = enable_sgm_single_dir ? sgm_single_mode : dir;
    Param param(n_dir, batch, n_cv, height, width, n_disp, dir_actual, rho, is_backward, is_training);
    param.enable_min_a_dir = enable_min_a_dir;
    UpdateParam(&param);
    param.dir = enable_sgm_single_dir ? 0 : param.dir;
    param.dir_inv = enable_sgm_single_dir ? 1 : param.dir_inv;
    param_list.push_back(param);
  }

  // TestLoop<<<n_block_unary, n_thread_unary>>>(param_list[0], msg_min_size, msg_update_ptr);

  for (uint iter = 0; iter < n_iter; ++iter) {
    if (is_training) {
      for (uint dir = 0; dir < n_dir; ++dir) {
        msg_min_index_address[dir] = msg_min_index_ptr + iter * msg_min_index_size + dir * msg_min_size;
        msg_norm_index_address[dir] = msg_norm_index_ptr + iter * msg_norm_index_size + dir * msg_norm_size;
      }
    }

    // Horizontal
    uint n_dir_hor = enable_sgm_single_dir ? 2 : min(2, n_dir);
    for (uint idx = 0; idx < n_dir_hor; ++idx) {
      if (enable_sgm_single_dir && idx != sgm_single_mode) continue;
      uint dir = enable_sgm_single_dir ? 0 : idx;
      UpdateUnaryKernel<<<n_block_unary, n_thread_unary>>>(param_list[dir],
                                                           msg_min_size,
                                                           unary_ptr,
                                                           msg_ptr,
                                                           unary_update_ptr);

#ifdef CUDA_ERROR_CHECK
      CUDAErrorCheck();
#endif

      uint n_threads = batch * n_cv * param_list[dir].n_trees * n_thread_a_tree;
      uint n_blocks = GetNumBlock(n_threads, n_thread_a_tree);
      HorizontalKernel<<<n_blocks, n_thread_a_tree>>>(param_list[dir],
                                                      n_threads,
                                                      n_thread_a_tree,
                                                      unary_update_ptr,
                                                      context_ptr,
                                                      edge_weight_address[dir],
                                                      msg_ptr,
                                                      msg_update_address[dir],
                                                      msg_min_index_address[dir],
                                                      msg_norm_index_address[dir]);

#ifdef CUDA_ERROR_CHECK
      CUDAErrorCheck();
#endif
    }

    // Vertical
    uint n_dir_ver = enable_sgm_single_dir ? 4 : min(4, n_dir);
    for (uint idx = 2; idx < n_dir_ver; ++idx) {
      if (enable_sgm_single_dir && idx != sgm_single_mode) continue;
      uint dir = enable_sgm_single_dir ? 0 : idx;
      UpdateUnaryKernel<<<n_block_unary, n_thread_unary>>>(param_list[dir],
                                                           msg_min_size,
                                                           unary_ptr,
                                                           msg_ptr,
                                                           unary_update_ptr);

#ifdef CUDA_ERROR_CHECK
      CUDAErrorCheck();
#endif

      uint n_threads = batch * n_cv * param_list[dir].n_trees * n_thread_a_tree;
      uint n_blocks = GetNumBlock(n_threads, n_thread_a_tree);
      DiagonalKernelWide<<<n_blocks, n_thread_a_tree>>>(param_list[dir],
                                                        n_threads,
                                                        n_thread_a_tree,
                                                        unary_update_ptr,
                                                        context_ptr,
                                                        edge_weight_address[dir],
                                                        msg_ptr,
                                                        msg_update_address[dir],
                                                        msg_min_index_address[dir],
                                                        msg_norm_index_address[dir]);

#ifdef CUDA_ERROR_CHECK
      CUDAErrorCheck();
#endif
    }

    // Diagonal
    uint n_dir_dia = enable_sgm_single_dir ? 16 : min(16, n_dir);
    for (uint idx = 4; idx < n_dir_dia; ++idx) {
      if (enable_sgm_single_dir && idx != sgm_single_mode) continue;
      uint dir = enable_sgm_single_dir ? 0 : idx;
      UpdateUnaryKernel<<<n_block_unary, n_thread_unary>>>(param_list[dir],
                                                           msg_min_size,
                                                           unary_ptr,
                                                           msg_ptr,
                                                           unary_update_ptr);

#ifdef CUDA_ERROR_CHECK
      CUDAErrorCheck();
#endif

      uint h_step_abs = std::abs(param_list[dir].h_step), w_step_abs = std::abs(param_list[dir].w_step);
      uint n_threads = batch * n_cv * param_list[dir].n_trees * n_thread_a_tree;
      uint n_blocks = GetNumBlock(n_threads, n_thread_a_tree);

      if (h_step_abs > w_step_abs)
        DiagonalKernelNarrow<<<n_blocks, n_thread_a_tree>>>(param_list[dir],
                                                            n_threads,
                                                            n_thread_a_tree,
                                                            unary_update_ptr,
                                                            context_ptr,
                                                            edge_weight_address[dir],
                                                            msg_ptr,
                                                            msg_update_address[dir],
                                                            msg_min_index_address[dir],
                                                            msg_norm_index_address[dir]);
      else
        DiagonalKernelWide<<<n_blocks, n_thread_a_tree>>>(param_list[dir],
                                                          n_threads,
                                                          n_thread_a_tree,
                                                          unary_update_ptr,
                                                          context_ptr,
                                                          edge_weight_address[dir],
                                                          msg_ptr,
                                                          msg_update_address[dir],
                                                          msg_min_index_address[dir],
                                                          msg_norm_index_address[dir]);

#ifdef CUDA_ERROR_CHECK
      CUDAErrorCheck();
#endif
    }

    size_t copy_size = msg_min_index_size * sizeof(float);
    hipMemcpy(msg_ptr, msg_update_ptr, copy_size, hipMemcpyDeviceToDevice);

    if (enable_cal_label) {
      hipMemcpy(cost_final_ptr, unary_ptr, msg_min_size * sizeof(float), hipMemcpyDeviceToDevice);
      CostAggregateKernel<<<n_block_unary, n_thread_unary>>>(enable_sgm,
                                                             param_list[0],
                                                             msg_min_size,
                                                             msg_update_ptr,
                                                             cost_final_ptr);

#ifdef CUDA_ERROR_CHECK
      CUDAErrorCheck();
#endif

      uint n_thread_label = min(MAX_THREADS_PER_BLOCK, msg_norm_size);
      uint n_block_label= (msg_norm_size + n_thread_label - 1) / n_thread_label;
      CalLabelKernel<<<n_block_label, n_thread_label>>>(param_list[0],
                                                        msg_norm_size,
                                                        cost_final_ptr,
                                                        label_all_ptr + iter * msg_norm_size);

#ifdef CUDA_ERROR_CHECK
      CUDAErrorCheck();
#endif
    }
  }

  hipMemcpy(cost_final_ptr, unary_ptr, msg_min_size * sizeof(float), hipMemcpyDeviceToDevice);
  CostAggregateKernel<<<n_block_unary, n_thread_unary>>>(enable_sgm,
                                                         param_list[0],
                                                         msg_min_size,
                                                         msg_update_ptr,
                                                         cost_final_ptr);
#ifdef CUDA_ERROR_CHECK
  CUDAErrorCheck();
#endif

  for (uint dir = 0; dir < n_dir; ++dir) {
    if (msg_update_address[dir] != nullptr) msg_update_address[dir] = nullptr;
    if (msg_min_index_address[dir] != nullptr) msg_min_index_address[dir] = nullptr;
    if (msg_norm_index_address[dir] != nullptr) msg_norm_index_address[dir] = nullptr;
    if (edge_weight_address[dir] != nullptr) edge_weight_address[dir] = nullptr;
  }
}

// Test MsgNorm function
__global__ void MsgNormTestWrap(const uint n_disp,
                                float* value) {
  uint tid = blockIdx.x * blockDim.x + threadIdx.x;
  uint current_d = tid % n_disp;
  if (current_d >= n_disp) return;

  uchar min_idx = 0;
  MsgNorm(n_disp, current_d, value, &min_idx);
  if (current_d == 0) printf("MsgNormTest value: %f, idx: %d.\n", value[0], min_idx);
}

void TestMsgNormCUDA(at::Tensor msg_norm) {
  const uint n_disp = msg_norm.size(1);
  float* value_ptr = msg_norm.data<float>();
  MsgNormTestWrap<<<1, n_disp>>>(n_disp, value_ptr);
}

// Test MultiStream function
// example from https://devblogs.nvidia.com/gpu-pro-tip-cuda-7-streams-simplify-concurrency
__global__ void StreamKernel(float* data) {
  uint tid = blockIdx.x * blockDim.x + threadIdx.x;
  data[tid] = tid;
}

void TestMultiStreamCUDA(const int enable_multiple,
                          at::Tensor data) {
  const uint n_dir = data.size(0);
  const uint n_element = data.size(1);
  float* data_ptr = data.data<float>();
  int n_threads = min(MAX_THREADS_PER_BLOCK, n_element);
  int n_blocks = (n_element + n_threads - 1) / n_threads;;
  hipStream_t streams[n_dir];

  for (uint dir = 0; dir < n_dir; ++dir) {
    if (enable_multiple == 1) {
      hipStreamCreate(&streams[dir]);
      StreamKernel<<<n_blocks, n_threads, 0, streams[dir]>>>(data_ptr + dir * n_element);
    } else {
      StreamKernel<<<n_blocks, n_threads, 0>>>(data_ptr + dir * n_element);
    }
  }
}

#ifdef __cplusplus
  }
#endif
